#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <stdlib.h>

void initWith(float num, float *a, int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = num;
    }
}

_global_ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = start; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}
void addVectors(float *result, float *a, float *b, int N)
{
    for (int i = 0; i < N; ++i)
    {
        result[i] = a[i] + b[i];
    }
}

int main()
{
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    float *ah;
    float *bh;
    float *ch;

    ah = (float *)malloc(size);
    bh = (float *)malloc(size);
    ch = (float *)malloc(size);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    initWith(3, ah, N);
    initWith(4, bh, N);
    initWith(0, ch, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    addVectorsInto<<<100, 1024>>>(c, a, b, N);
    // hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    clock_t begin = clock();
    addVectors(ch, ah, bh, N);
    clock_t end = clock();
    double time_spent = (double)1000 * (end - begin) / CLOCKS_PER_SEC;

    printf("GPU time= %f ms\n", et);

    printf("CPU time= %lf ms\n", time_spent);

    // checkElementsAre(7, c, N);

    free(ah);
    free(bh);
    free(ch);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}